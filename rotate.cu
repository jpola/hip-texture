#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "image_descriptor.hpp"
#include <stdio.h>

// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

template<typename T>
__global__ void transformKernel(T* outputData,
                                int width,
                                int height,
                                T theta)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    T u = (T)x - (T)width/2;
    T v = (T)y - (T)height/2;
    T tu = u*cosf(theta) - v*sinf(theta);
    T tv = v*cosf(theta) + u*sinf(theta);

    tu /= (T)width;
    tv /= (T)height;

    // read from texture and write to global memory
    T val = tex2D(tex, tu+0.5f, tv+0.5f);
    if(x == 0 && y == 0)
        printf("u = %f, v = %f, tu = %f, tv = %f, val = %f\n", u, v, tu, tv, val);
    if(x == 1 && y == 0)
        printf("u = %f, v = %f, tu = %f, tv = %f, val = %f\n", u, v, tu, tv, val);
    if(x == 2 && y == 0)
        printf("u = %f, v = %f, tu = %f, tv = %f, val = %f\n", u, v, tu, tv, val);
    outputData[y*width + x] = val;
}


template<typename T>
void transform_image(T* data, const image_description<T>& img_desc, T angle)
{

    size_t size = img_desc.width * img_desc.height * sizeof(T);

    T* d_data = NULL;
    hipMalloc((void **) &d_data, size);

    /**
     * Example, for float texels we could create a channel with
     *
     * hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
     *
     * while for short4 texels this would be
     *
     * hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindSigned);
     */
    //define channel descriptor for image 32 means 32 bits per pixel = float;
    hipChannelFormatDesc channelDesc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    //prepare array to fetch data from. It is the copy of input image
    hipArray *cuArray;
    hipMallocArray(&cuArray, &channelDesc, img_desc.width, img_desc.height);
    hipMemcpyToArray(cuArray, 0, 0, data, size, hipMemcpyHostToDevice);

    //set texture parameters
    if(img_desc.x_addr_mode == CLAMP)
    {
        tex.addressMode[0] = hipAddressModeWrap;
        tex.addressMode[1] = hipAddressModeClamp;
    }
    else
    {
        tex.addressMode[0] = hipAddressModeWrap;
        tex.addressMode[1] = hipAddressModeWrap;

    }
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    hipBindTextureToArray(tex, cuArray, channelDesc);

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(img_desc.width / dimBlock.x, img_desc.height / dimBlock.y, 1);

    transformKernel<<<dimGrid, dimBlock, 0>>>(d_data,
                                              (int)img_desc.width,
                                              (int)img_desc.height,
                                              angle);

    hipDeviceSynchronize();
    hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFreeArray(cuArray);
}

//instantiation of a template definition
// other option is to move the definition to header file
template void transform_image<float>(float* data, const image_description<float>& img_desc, float f);
